#include <iostream>
#include <hip/hip_runtime.h>

// Função de verificação de erro CUDA
inline void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int N = 10000;  // Tamanho dos vetores
    int *a, *b, *c;  // Vetores de entrada e saída
    int *d_a, *d_b, *d_c;  // Ponteiros para a memória da GPU

    // Aloca memória para os vetores no host
    a = new int[N];
    b = new int[N];
    c = new int[N];

    // Inicializa os vetores
    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Aloca memória na GPU
    hipMalloc(&d_a, N * sizeof(int));
    checkCudaError("Erro ao alocar memória para d_a");

    hipMalloc(&d_b, N * sizeof(int));
    checkCudaError("Erro ao alocar memória para d_b");

    hipMalloc(&d_c, N * sizeof(int));
    checkCudaError("Erro ao alocar memória para d_c");

    // Copia os dados dos vetores para a memória da GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError("Erro ao copiar dados para d_a");

    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError("Erro ao copiar dados para d_b");

    // Define o número de threads por bloco e o número de blocos
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Chama o kernel CUDA
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    checkCudaError("Erro ao lançar o kernel");

    // Copia o resultado de volta para a memória do host
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    checkCudaError("Erro ao copiar dados de volta para c");

    // Imprime todos os valores de a, b e c
	std::cout << "Valores sendo somados e o resultado (a + b = c):" << std::endl;
	for (int i = 0; i < N; ++i) {
		std::cout << "a[" << i << "] = " << a[i] << " + b[" << i << "] = " << b[i] << " => c[" << i << "] = " << c[i] << std::endl;
	}


    // Verifica o resultado
    for (int i = 0; i < N; ++i) {
        if (c[i] != a[i] + b[i]) {
            std::cout << "Erro no calculo! Indice: " << i << std::endl;
            break;
        }
    }

    // Libera a memória
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] a;
    delete[] b;
    delete[] c;

    std::cout << "Somador de vetores concluido!" << std::endl;

    return 0;
}
